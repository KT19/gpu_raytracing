#ifndef ERROR_HANDLING_CU
#define ERROR_HANDLING_CU
#include<iostream>

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if(result) {
        std::cerr<<"CUDA error = "<<static_cast<unsigned int>(result)<<" at "<<
        file<<":"<<line<<" '"<<func<<" '\n";
        hipDeviceReset();
        exit(-1);
    }
}

#endif